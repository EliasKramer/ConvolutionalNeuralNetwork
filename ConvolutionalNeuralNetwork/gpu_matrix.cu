#include "gpu_matrix.cuh"

void gpu_matrix::check_for_valid_args()
{
	if (width == 0 || height == 0 || depth == 0)
	{
		throw std::invalid_argument("could not create gpu_matrix");
	}
}

void gpu_matrix::check_for_last_cuda_error()
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(err));
	}
}

void gpu_matrix::free_owned_gpu_mem()
{
	if (owns_gpu_mem_ptr)
	{
		hipFree(gpu_ptr);
	}
	owns_gpu_mem_ptr = false;
}

gpu_matrix::gpu_matrix(
	size_t width,
	size_t height,
	size_t depth
) :
	width(width),
	height(height),
	depth(depth),
	owns_gpu_mem_ptr(true)
{
	check_for_valid_args();
	hipMalloc(&gpu_ptr, item_count() * sizeof(float));
	check_for_last_cuda_error();
}

gpu_matrix::gpu_matrix(
	float* given_gpu_ptr,
	size_t width,
	size_t height,
	size_t depth
) :
	gpu_ptr(given_gpu_ptr),
	width(width),
	height(height),
	depth(depth),
	owns_gpu_mem_ptr(false)
{
	check_for_valid_args();
	//TODO if there is a way to check how much is allocated on the given ptr, 
	//then check if that matches the given height, width and depth
}

gpu_matrix::gpu_matrix(const matrix& m, bool copy_values)
	:gpu_matrix(m.get_width(), m.get_height(), m.get_depth())
{
	if (copy_values)
	{
		set_values(m);
	}
}

gpu_matrix::~gpu_matrix()
{
	free_owned_gpu_mem();
}

const float* gpu_matrix::get_gpu_memory_readonly() const
{
	return gpu_ptr;
}

float* gpu_matrix::get_gpu_memory()
{
	return gpu_ptr;
}

size_t gpu_matrix::get_width() const
{
	return width;
}

size_t gpu_matrix::get_height() const
{
	return height;
}

size_t gpu_matrix::get_depth() const
{
	return depth;
}

void gpu_matrix::set_values(const matrix& m)
{
	if (m.flat_readonly().size() != item_count())
	{
		throw std::runtime_error("gpu_memory size mismatch");
	}

	hipMemcpy(
		gpu_ptr,
		m.flat_readonly().data(),
		item_count() * sizeof(float),
		hipMemcpyHostToDevice);

	check_for_last_cuda_error();
}

void gpu_matrix::set_all(float value)
{
	std::vector<float> values(item_count(), value);
	hipMemcpy(
		gpu_ptr,
		values.data(),
		item_count() * sizeof(float),
		hipMemcpyHostToDevice);
	check_for_last_cuda_error();
}

std::unique_ptr<matrix> gpu_matrix::to_cpu() const
{
	//TOOOOOOOOOOOOOOOOOOOOODOOOOOOOOOOOOOOOOOo
	return std::unique_ptr<matrix>();
}

float* gpu_matrix::get_gpu_ptr_layer(size_t depth_idx)
{
	return sub_ptr<float>(gpu_ptr, width * height, depth_idx);
}

float* gpu_matrix::get_gpu_ptr_row(size_t height_idx, size_t depth_idx)
{
	return get_gpu_ptr_layer(depth_idx) + height_idx * width;
}

float* gpu_matrix::get_gpu_ptr_item(size_t width_idx, size_t height_idx, size_t depth_idx)
{
	return get_gpu_ptr_row(height_idx, depth_idx) + width_idx;
}

size_t gpu_matrix::item_count() const
{
	return width * height * depth;
}