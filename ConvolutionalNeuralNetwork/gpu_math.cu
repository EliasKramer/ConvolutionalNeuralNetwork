#include "hip/hip_runtime.h"
#include "gpu_math.cuh"

#define THREADS_PER_BLOCK 1024

static unsigned int get_block_count(unsigned int size)
{
	//if we have 1024 elements, we need 1 block
	//if we have 1025 elements, we need 2 blocks
	//if we have 2048 elements, we need 2 blocks
	//and as long as it is under 1024 - 1 thread will still work
	return ((size - 1) / THREADS_PER_BLOCK) + 1;
}

static void set_device()
{
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		throw std::runtime_error("hipSetDevice failed " + cudaStatus);
	}
}

static void check_for_error_and_synchronize()
{
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::string cuda_status = hipGetErrorString(cudaStatus);
		throw std::runtime_error("error while executing cuda kernel cuda status:" + cuda_status);
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		std::string cuda_status = hipGetErrorString(cudaStatus);
		throw std::runtime_error("could not sync cuda device cuda status:" + cuda_status);
	}
}

__device__ int get_idx(int x, int y, int z, int height, int width)
{
	return x + y * width + z * width * height;
}

__global__ void gpu_dot_product_kernel(
	const float* weights,
	const float* input,
	const int input_size,
	float* activations,
	const int activations_size)
{
	unsigned int activation_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (activation_idx < activations_size)
	{
		float sum = 0;
		for (int i = 0; i < input_size; i++)
		{
			int weight_idx = get_idx(i, activation_idx, 0,
				activations_size, input_size);
			sum += weights[weight_idx] * input[i];
		}
		activations[activation_idx] = sum;
	}
}

float* gpu_sub_ptr(gpu_memory<float>& gpu_memory, size_t elements_per_idx, size_t index)
{
	if (gpu_memory.gpu_data_ptr() == nullptr)
	{
		throw std::invalid_argument("gpu_sub_ptr failed. gpu_memory.gpu_data_ptr() is null");
	}
	if (index >= gpu_memory.item_count())
	{
		throw std::invalid_argument("gpu_sub_ptr failed. index out of range");
	}
	if (gpu_memory.size() < elements_per_idx * sizeof(float))
	{
		throw std::invalid_argument("gpu_sub_ptr failed. size_of_element must be less than gpu_memory.size()");
	}

	return (float*)((char*)gpu_memory.gpu_data_ptr() + index * elements_per_idx * sizeof(float));
}

void gpu_dot_product(
	const gpu_memory<float>& gpu_weights,
	const gpu_memory<float>& gpu_input,
	gpu_memory<float>& gpu_activations)
{
	if (gpu_weights.item_count() == 0 ||
		gpu_input.item_count() == 0 ||
		gpu_activations.item_count() == 0)
	{
		throw std::invalid_argument("gpu_dot_product failed. size must be greater than 0");
	}
	if (gpu_activations.item_count() * gpu_input.item_count() != gpu_weights.item_count())
	{
		throw std::invalid_argument("gpu_dot_product failed. false format");
	}

	set_device();

	unsigned int size = gpu_activations.item_count();
	unsigned int block_count = get_block_count(size);
	gpu_dot_product_kernel << < block_count, THREADS_PER_BLOCK >> > (
		gpu_weights.gpu_data_ptr(),
		gpu_input.gpu_data_ptr(),
		gpu_input.item_count(),
		gpu_activations.gpu_data_ptr(),
		gpu_activations.item_count());

	check_for_error_and_synchronize();
}

__global__ void gpu_add_matrices_kernel(const float* a, const float* b, float* result, unsigned int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
	{
		result[index] = a[index] + b[index];
	}
}
void gpu_add(
	const gpu_memory<float>& gpu_memory_a,
	const gpu_memory<float>& gpu_memory_b,
	gpu_memory<float>& gpu_memory_result)
{
	if (gpu_memory_a.item_count() == 0 ||
		gpu_memory_a.item_count() != gpu_memory_b.item_count() ||
		gpu_memory_a.item_count() != gpu_memory_result.item_count())
	{
		throw std::invalid_argument("gpu_add_matrices failed. size must be greater than 0");
	}

	set_device();

	unsigned int size = gpu_memory_a.item_count();

	gpu_add_matrices_kernel << < get_block_count(size), THREADS_PER_BLOCK >> > (
		gpu_memory_a.gpu_data_ptr(),
		gpu_memory_b.gpu_data_ptr(),
		gpu_memory_result.gpu_data_ptr(),
		size);

	check_for_error_and_synchronize();
}

__global__ void gpu_valid_cross_correlation_kernel(
	const float* input,
	const float* weights,
	float* result,
	const int input_depth,
	const int input_width,
	const int kernel_width,
	const int output_width,
	const int stride)
{
	unsigned int result_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (result_idx < output_width * output_width)
	{
		//print all arguments
		printf("input_depth: %d, input_width: %d, kernel_width: %d, output_width: %d, stride: %d\n",
			input_depth, input_width, kernel_width, output_width, stride);

		int input_x = (result_idx % output_width) * stride;
		int input_y = (result_idx / output_width) * stride;

		float sum = 0;
		for (int kernel_x = 0; kernel_x < kernel_width; kernel_x++)
		{
			for (int kernel_y = 0; kernel_y < kernel_width; kernel_y++)
			{
				for (int kernel_z = 0; kernel_z < input_depth; kernel_z++)
				{
					int input_idx = get_idx(input_x + kernel_x, input_y + kernel_y, kernel_z, input_width, input_width);
					int weight_idx = get_idx(kernel_x, kernel_y, kernel_z, kernel_width, kernel_width);
					sum += input[input_idx] * weights[weight_idx];
				}
			}
		}
		printf("result: %f\n", sum);
		result[result_idx] = sum;
	}
}

void gpu_valid_cross_correlation(
	const gpu_memory<float>& gpu_input,
	const std::vector<gpu_memory<float>>& gpu_kernel_weights,
	gpu_memory<float>& gpu_activations,
	size_t input_width,
	size_t input_depth,
	size_t kernel_width,
	size_t kernel_count,
	size_t stride,
	size_t output_width)
{
	if (gpu_input.item_count() == 0 ||
		gpu_activations.item_count() == 0 ||
		gpu_kernel_weights.size() == 0)
	{
		throw std::invalid_argument("gpu_valid_cross_correlation failed. size must be greater than 0");
	}
	if (input_width * input_width * input_depth != gpu_input.item_count())
	{
		throw std::invalid_argument("input size is different on gpu and cpu");
	}
	if (!is_whole_number(gpu_kernel_weights[0].item_count() / kernel_count))
	{
		throw std::invalid_argument("gpu kernels could not be devided by kernel count");
	}
	if (kernel_width * kernel_width * input_depth != (gpu_kernel_weights[0].item_count() / kernel_count))
	{
		throw std::invalid_argument("kernel size false");
	}

	const float output_side_size = (input_width - kernel_width) / (float)stride + 1;
	if (!is_whole_number(output_side_size))
	{
		throw std::invalid_argument("gpu_valid_cross_correlation failed. this stride, size combination cannot be used");
	}
	if (gpu_activations.item_count() != output_side_size * output_side_size * kernel_count)
	{
		throw std::invalid_argument("gpu_valid_cross_correlation failed. false format");
	}

	set_device();

	for (int activation_depth = 0; activation_depth < kernel_count; activation_depth++)
	{
		//splits the gpu_activations into its dephts
		float* activation_ptr = gpu_sub_ptr(gpu_activations, output_width * output_width, activation_depth);

		size_t block_count = get_block_count(output_width * output_width);

		gpu_valid_cross_correlation_kernel << <(int)block_count, THREADS_PER_BLOCK >> > (
			gpu_input.gpu_data_ptr(),
			gpu_kernel_weights[activation_depth].gpu_data_ptr(),
			activation_ptr,
			(int)input_depth,
			(int)input_width,
			(int)kernel_width,
			(int)output_width,
			(int)stride);
	}
	check_for_error_and_synchronize();
}

__global__ void gpu_sigmoid_kernel(float* data, int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
	{
		data[index] = 1 / (1 + exp(-data[index]));
	}
}

void gpu_sigmoid(gpu_memory<float>& gpu_memory)
{
	if (gpu_memory.item_count() == 0)
	{
		throw std::invalid_argument("gpu_sigmoid failed. size must be greater than 0");
	}

	set_device();

	unsigned int size = gpu_memory.item_count();
	gpu_sigmoid_kernel << < get_block_count(size), THREADS_PER_BLOCK >> > (
		gpu_memory.gpu_data_ptr(),
		size);

	check_for_error_and_synchronize();
}

__global__ void gpu_relu_kernel(float* data, int size)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size)
	{
		data[index] = data[index] > 0 ? data[index] : 0;
	}
}

void gpu_relu(gpu_memory<float>& gpu_memory)
{
	if (gpu_memory.item_count() == 0)
	{
		throw std::invalid_argument("gpu_relu failed. size must be greater than 0");
	}

	set_device();

	unsigned int size = gpu_memory.item_count();
	gpu_relu_kernel << < get_block_count(size), THREADS_PER_BLOCK >> > (
		gpu_memory.gpu_data_ptr(),
		size);

	check_for_error_and_synchronize();
}