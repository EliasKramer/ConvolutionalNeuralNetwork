#include "gpu_nn_data_block.cuh"

size_t gpu_nn_data_block::elements_in_block() const
{
	return (num_of_data + num_of_label_data);
}

gpu_nn_data_block::gpu_nn_data_block(
	size_t num_of_blocks,
	size_t num_of_data,
	size_t num_of_label_data) :
	num_of_blocks(num_of_blocks),
	num_of_label_data(num_of_label_data),
	num_of_data(num_of_data)
{
	if (num_of_blocks == 0 || num_of_data == 0)
		throw std::runtime_error("could not create gpu_nn_data_block");

	data = std::make_unique<gpu_memory<float>>(num_of_blocks * elements_in_block());
}

size_t gpu_nn_data_block::get_num_of_blocks() const
{
	return num_of_blocks;
}

size_t gpu_nn_data_block::get_num_of_label_data() const
{
	return num_of_label_data;
}

size_t gpu_nn_data_block::get_num_of_data() const
{
	return num_of_data;
}

float* gpu_nn_data_block::get_gpu_data_ptr(int idx)
{
	if (idx >= num_of_blocks || idx < 0)
		throw std::runtime_error("index out of bounds");

	return gpu_sub_ptr(*data.get(), elements_in_block(), idx);
}

float* gpu_nn_data_block::get_gpu_label_ptr(int idx)
{
	if (idx >= num_of_blocks || idx < 0)
		throw std::runtime_error("index out of bounds");
	if (num_of_label_data == 0)
		throw std::runtime_error("this block has no label data");

	return gpu_sub_ptr(*data.get(), elements_in_block(), idx) + (num_of_data);
}

void gpu_nn_data_block::set_data(int idx, const std::vector<float>& data)
{
	if (data.size() != num_of_data)
		throw std::runtime_error("could net set gpu data in block");

	hipMemcpy(get_gpu_data_ptr(idx), data.data(), data.size() * sizeof(float), hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess)
		throw std::runtime_error("could net set gpu data in block");
}

void gpu_nn_data_block::set_label_data(int idx, const std::vector<float>& data)
{
	if (data.size() != num_of_label_data)
		throw std::runtime_error("could net set gpu data in block");

	hipMemcpy(get_gpu_label_ptr(idx), data.data(), data.size() * sizeof(float), hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess)
		throw std::runtime_error("could net set gpu data in block");
}

void gpu_nn_data_block::set_data(
	std::vector<nn_data>::const_iterator begin,
	std::vector<nn_data>::const_iterator end)
{
	if (std::distance(begin, end) != num_of_blocks ||
		begin[0].get_data().flat_readonly().size() != num_of_data ||
		begin[0].get_label().flat_readonly().size() != num_of_label_data)
		throw std::runtime_error("could net set gpu data in block");

	std::vector<float> tmp = std::vector<float>(num_of_blocks * elements_in_block());

	int count = 0;
	for (auto it = begin; it != end; ++it)
	{
		std::copy(
			it->get_data().flat_readonly().begin(), 
			it->get_data().flat_readonly().end(), 
			tmp.begin() + (count * elements_in_block()));
		std::copy(
			it->get_label().flat_readonly().begin(),
			it->get_label().flat_readonly().end(),
			tmp.begin() +  (count * elements_in_block()) + num_of_data);

		count++;
	}

	hipMemcpy(data.get()->gpu_data_ptr(), tmp.data(), tmp.size() * sizeof(float), hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess)
		throw std::runtime_error("could net set gpu data in block");
}
